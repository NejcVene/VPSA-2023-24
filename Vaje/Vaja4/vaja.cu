#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// tole rabimo za CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N (size_t) (1024 * 1024 * 1024) // 1B elementov
#define T (size_t) (1024 * 1024 * 2) // 2M niti

// bs (block size = 128, 256, 512, 1024)
// T = 2^16 (1024 * 64), 2^17, 2^18, 2^19, 2^20
// 20 zagonov skupaj za zgornje parametre
// probi ugotovit, kateri par da najboljše rezultate
// mores tko 3-krat, 4-krat zagnat da dobiš "realen (al kak bi reku)" rezultat

#define BS 1024
#define GS (T / BS)

// size_t = unsigned long (64 bit)

/*
Sedaj imamo problem, ker smo dosegli max. število niti. Kako to odpraviti?
    Ko nit zaključi, se mora njen index predstaviti za toliko, da začne delati
    na drugem bloku. Posledično vsaka nit naredi malo več dela.
*/

// how to run:
// 1. module load CUDA
// 2. nvcc -o program program.cu
// 3. srun --partition=gpu --gpus=1 --ntasks=1 (--mem-per-cpu=4GB tole naj bi povečal kolk spomina mamo na voljo) program

/*
program bo sestavljen iz dveh funkcij, in sicer:
    - funkcija za gostitelja npr. main (prevede se s prevajalnikom za gostitelja)
    - najmanj ene funkcije, ki se izvaja na GPU (prevede se s prevajalnikom za GPU)-
*/

/*
koda znotraj main ne bo videla podatkov, operadnov na napravi (in obratno).
GPU NE MORE vrniti vrednost registru, ki bi ga prebrala CPU.
Skratka ne more vrniti ničesa, zato mora biti void.

Da prevajalnik loči te naše funkcije, moramo tiste za GPU označiti:
    __global__ -> izvaja, prevede na GPU in jo začene CPU
    __local__  -> prevede, izvede in začene na GPU
*/

// če je možno začenemo toliko niti, kolikor je dolg vektor.
// če je pa več niti kot je dolg vektor, je treba te odvečne niti ustaviti oz. preprečiti
// da ne preberejo spomina, ki ga ne uporabljamo.
// zato smo napisali ta if stavek:
//  if (tid < n) {
//    ...
//  }

// koda na napravi (ščepec oz. kernel)
// __gobal__ pove, da se izvaja na gostitelju (gcc jo bo ignoriral)
__global__ void sestejVektorje(const float *a, const float *b, float *delniSkalarniProdukt, const size_t n) {
    /*
    a, b, c naslovi posameznik vektorjev
    n -> koliko elementov je v polju

    Zakaj const? Onačuje, da kazalec kaže na nekaj, kar bo shranjeno v pomn. konstant.
    Const ne bo dopuščal spreminjanje vrednosti.
BS
    Potrebno je še ugotoviti, kdo sem in katere elemente lahko seštevam.
    */

    // indeks vektoraja = indeks bloka * velikost + indeks niti v bloku
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x; // tole so vgrajene spremenjivke okolja CUDA in jih lahko beremo kadarkoli med izvajanjem

    __shared__ float lokalniDelniProdukti[BS];

    while (tid < n) { // če je indeks manjši od dolžine vektorja, smemo delati naslednjo operacijo. Če je večji ne smemo it, saj tam ni nič.
        [tid] = a[tid] + b[tid]; // vsaka nit sešteje dva istoležna element a v vektorju 
        tid += blockDim.x * gridDim.x; // tole sedaj prestavi nit v drugi blok na isti index
    }

    __syncthreads(); // tale funkcija je prepreka

    if (tid == 0) { // nit s indexom 0
        for (size_t i = 0; i<n; i++) {
            dp += c[i];
        }
    }

}

// koda na gostitelju
// gostitelj je vedno zadolžen za pripravo podatkov, ki se bodo nato uporabili
size_t main(void) {

    size_t dataSize = N * sizeof(float);

    /*
    1. malloc za vektorje a, b in c na gostitelju
    1.1 init vektorjev a in b
    2. malloc za vektorje a, b in c na napravi
    3. kopiraj vektorja a in b na napravo (s uporabo gonilnika, ne load/store ukazi)
    4. zaženi ščepec na napravi
    5. ko se ščepec zaključi, kopiraj rezultat s naprave na gostitelja
    6. preveri in izpiši rezultat
    7. počisti pomnilnik (gostitelja in naprave)
    */

    // 1
    float *hVecA = (float *) malloc(dataSize);
    float *hVecB = (float *) malloc(dataSize);
    float *hVecC = (float *) malloc(dataSize);

    // 1.1
    for (size_t i = 0; i<N i++) {
        hVecA[i] = 1.0;
        hVecB[i] = 2.0;
    }

    // 2
    // Tega me more naredit malloc, zato uporabimo od CUDA
    // uporabi funckijo hipMalloc((naslov pomnilniške besede na napravi))
    // dobra praksa je, da preberimo kaj nam tale funkcija vrne (če se je uspešno izvedla)
    
    float *dVecA;
    float *dVecB;
    float *dVecC;
    hipMalloc(&dVecA, dataSize);
    hipMalloc(&dVecB, dataSize);
    hipMalloc(&dVecC, dataSize);

    // 3
    // prosi DMA, da te vrednosti prenese
    // uporabi funkcijo hipMemcpy(void *dst (naprava), vodi *scr (gostitelj), size_t count (koliko bajtov kopirat), cudaMemcpyToDevice (smer prenosa));

    hipMemcpy(dVecA, hvecA, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(dVecB, hvecB, dataSize, hipMemcpyHostToDevice);

    // 4
    // pokliči funkcijo, ki se zaganja na napravi
    // in pred/ob zagonu definiraj izvajalno okole (koliko blokov v mreži (gridSize), koliko niti v bloku (blokSize))

    // dim3 je struktura
    size_t bs = 1024; // block size
    dim3 blockSize(bs, 1, 1); // 256 niti v bloku (1024 je max. število niti v bloku!)
    dim3 gridSize(T / bs, 1, 1); // en blok v mreži
    sestejVektorje<<<gridSize, blockSize>>>(dvecA, dVecB, dVecC, N);
    // ^ teti <<<>>> podajo izvajalno okolje

    // 5

    // problematično 
    hipMemcpy(hVecC, dVecC, dataSize, hipMemcpyDeviceToHost);

    // 6
    // uporabi double, ker imamo 52 bitov za mantiso in je bolj natančen kot float
    double rezultat = 0.0;
    for (size_t i = 0; i<N; i++) {
        rezultat += (double) hVecA[i];
    }
    prsize_tf("Rezultat = %f\n", rezultat); // rezltat mora biti 3072

    // 7

    free(hVecA);
    free(hVecB);
    free(hVecC);

    hipFree(dVecA);
    hipFree(dVecB);
    hipFree(dVecC);

    return 0;

}

/*
Za merjenje časa:
    Ko se dogodek naredi, si zapomnimo ta čas.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start); // si znači ta dogodek, torej čas.
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    loat miliseconds = 0;
    hipEventElapsedTime(&milicedonds, start, stop);
    printf("Execution time: %0.3f\n", miliseconds);
*/